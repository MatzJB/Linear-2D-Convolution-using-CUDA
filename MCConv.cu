#include "hip/hip_runtime.h"

//Matz JB June 2012

//This code performs the convolution between two images
//IFFT( FFT2(A).*FFT2(B) ) where A:(MxN), B:(MxN)
//Memory requirement: 6 units

#include <stdio.h>
#include <math.h> 
#include <hipfft/hipfft.h>
#include "errCodes.h" 
//contains the hidden error message to mexprint, because we cannot print from the kernel

#define MAX_THREADS 1024 //the maximum number of threads for this GPU, change as appropriate


void sync()
{
hipDeviceSynchronize();
}

void cudaRelease(void *ptr) 
{
  if ( ptr != NULL ) 
    hipFree(ptr); 
}

//http://gpgpu.org/wp/wp-content/uploads/2009/06/03-Toolkit.pdf
//Standard way of interlacing/weaving using the kernel
__global__ void weavecomplex (hipfftComplex *c, float *a, int M, int N)
{
int idx = blockIdx.x*blockDim.x + threadIdx.x;
int idy = blockIdx.y*blockDim.y + threadIdx.y;

if(idx<M && idy<N)
{

int index = idx + idy*M;
c[index].x  = a[index];
c[index].y  = 0.f;

}
}


//We only need to unweave to real
__global__ void unweavecomplex2R (float *a, hipfftComplex *c, int M, int N)
{

int idx = blockIdx.x*blockDim.x + threadIdx.x;
int idy = blockIdx.y*blockDim.y + threadIdx.y;
volatile float2 c2;//force vector load, increase memory coalescing

if(idx<M && idy<N)
	{
	int index = idx + idy*M;

	c2.x = c[index].x;
	c2.y = c[index].y;
	a[index] = c2.x;
	}
}


//Scaling is embedded in Hadamard product instead of inside the "weaving" functions
__global__ void hadamard3(hipfftComplex * a, hipfftComplex * b, int M, int N)
{
int idx = blockIdx.x*blockDim.x + threadIdx.x;
int idy = blockIdx.y*blockDim.y + threadIdx.y;
float scaling = 1.0f/sqrt(1.0f*M*N);
int index;
volatile float tmp;

	if(idx<M && idy<N)
	{
		index = idx + idy*M;
	
		a[index].x *= scaling;
		a[index].y *= scaling;
		b[index].x *= scaling;
		b[index].y *= scaling;

		tmp = a[index].x;
	
	//Naive complex multiplication, 2 addition, 4 multiplication
		a[index].x = tmp*b[index].x - a[index].y*b[index].y;
		a[index].y = tmp*b[index].y + a[index].y*b[index].x;
	}
}



//Convolves a and b and store the result in c.
extern "C" void Convolution(float *a, float *b, float *c, int M, int N, int *err)
{
	//Device data, only used on the device:
	//These declarations must be first since we risk to encounter errors for which we just go to "Error"
	hipfftComplex *rhs_complex_d1 = NULL;
	hipfftComplex *rhs_complex_d2 = NULL;

	float *a_d = NULL;
	
//Setting up Block and Grids for the thread mappping:
  int block_size_x = 32; //MAX_THREADS=1024, sqrt(1024) = 32
  int block_size_y = block_size_x;

  dim3 dimBlock(block_size_x, block_size_y, 1);
  dim3 dimGrid((M/dimBlock.x), (N/dimBlock.y));

  if (M % block_size_x !=0) 
	dimGrid.x+=1;

  if (N % block_size_y !=0) 
	dimGrid.y+=1;
    
  hipfftHandle plan;
  *err = ERR_FAILSAFE;

//1 unit
	if( hipMalloc((void **) &a_d, sizeof(float)*M*N) != hipSuccess )
    {
		*err = ERR_MALLOC;
		goto Error;
    }
		
    if( hipMemcpy(a_d, a, sizeof(float)*M*N, hipMemcpyHostToDevice) != hipSuccess )
    {
      *err = ERR_COPY;
      goto Error;
    }

		//3 units
	if( hipMalloc((void **) &rhs_complex_d1, sizeof(hipfftComplex)*M*N) != hipSuccess )
    {
		*err = ERR_MALLOC;
		goto Error;
    }
	
	weavecomplex<<<dimGrid, dimBlock>>>(rhs_complex_d1, a_d, M, N);
	
	//5 units
	if( hipMalloc((void **) &rhs_complex_d2, sizeof(hipfftComplex)*M*N) != hipSuccess )
    {
		*err = ERR_MALLOC;
		goto Error;
    }
	//sync();//better?

	//reuse a_d
	 if( hipMemcpy(a_d, b, sizeof(float)*M*N, hipMemcpyHostToDevice) != hipSuccess )
    {
      *err = ERR_COPY;
      goto Error;
    }

	weavecomplex<<<dimGrid, dimBlock>>>(rhs_complex_d2, a_d, M, N);
	
	sync();//must wait for a_d to finish
	//4 units
	cudaRelease(a_d);

//At least 6 units
  if (hipfftPlan2d(&plan, N, M, HIPFFT_C2C) != HIPFFT_SUCCESS)
    {
	  *err = ERR_PLAN;
	  goto Error;//added this 26/5   
    }

	if (cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE) != HIPFFT_SUCCESS)
	//if (cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_FFTW_PADDING) != HIPFFT_SUCCESS)
	//if (cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_FFTW_ASYMMETRIC) != HIPFFT_SUCCESS)
	//if (cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_FFTW_ALL) != HIPFFT_SUCCESS)	
	{
		*err = ERR_COMPAT;
		goto Error;
    }
	

  if (hipfftExecC2C(plan, rhs_complex_d1, rhs_complex_d1, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
    {
		*err = ERR_FFT_FORWARD;
		goto Error;
    }

	//same plan to perform FFT on the other matrix
	if (hipfftExecC2C(plan, rhs_complex_d2, rhs_complex_d2, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
    {
		*err = ERR_FFT_FORWARD;
		goto Error;
    }

	hadamard3<<<dimGrid, dimBlock>>>(rhs_complex_d1, rhs_complex_d2, M, N);
//sync();

	if (hipfftExecC2C(plan, rhs_complex_d1, rhs_complex_d1, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
    {
		*err = ERR_FFT_INVERSE;
		
      goto Error;
    }
	
		cudaRelease(rhs_complex_d2);
	
		if( hipMalloc((void **) &a_d, sizeof(float)*M*N) != hipSuccess )
		{
		*err = ERR_MALLOC;
		goto Error;
		}
	
		unweavecomplex2R<<<dimGrid, dimBlock>>>(a_d, rhs_complex_d1, M, N);
	
	//Pick only real part and send back to host code	
	unweavecomplex2R<<<dimGrid, dimBlock>>>(a_d, rhs_complex_d1, M, N);
//sync(); //really?	
	hipMemcpy(c, a_d, sizeof(float)*M*N, hipMemcpyDeviceToHost);
		
	*err = ERR_OK;	//We reached this point thus everything went ok, otherwise we have ERR_FAILSAFESAFE, which should never happen

	
//Catch all error cases and clean up:	
	Error:
	
	cudaRelease(a_d);
	cudaRelease(rhs_complex_d1);
	
	hipfftDestroy(plan);
		}
