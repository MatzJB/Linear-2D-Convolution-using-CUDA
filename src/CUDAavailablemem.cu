//Matz JB Aug 2012
//This code only return the memory free on the device


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h> 
#include <hipfft/hipfft.h>

extern "C" void CUDAavailablemem(int *total, int *free)
{
	size_t memfree;
	size_t memtotal;

	hipMemGetInfo(&memfree, &memtotal);
	*total = memtotal/pow(2.0,20);
	*free = memfree/pow(2.0,20);
}
