//Matz JB June 2012
//This code only return the memory allocated by the plan


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h> 
#include <hipfft/hipfft.h>

extern "C" void fetchplanmem(int M, int N, int *planmemory, int *total)
{
	size_t memfree;
	size_t memtotal;
	size_t currentmem;

	hipfftHandle plan;

	hipMemGetInfo(&memfree, &memtotal);
	*total = memtotal/pow(2.0,20);
	
	hipfftPlan2d(&plan, N, M, HIPFFT_C2C);
    hipMemGetInfo(&currentmem, &memtotal);
	
    *planmemory = (int) (memfree - currentmem)/pow(2.0, 20);
	
	hipfftDestroy(plan);
}
